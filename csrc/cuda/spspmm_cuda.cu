#include "spspmm_cuda.h"

#include <ATen/cuda/HIPContext.h>
#include <hipsparse.h>

#include "utils.cuh"

#define AT_DISPATCH_CUSPARSE_TYPES(TYPE, ...)                                  \
  [&] {                                                                        \
    switch (TYPE) {                                                            \
    case torch::ScalarType::Float: {                                           \
      using scalar_t = float;                                                  \
      const auto &cusparsecsrgemm2_bufferSizeExt =                             \
          hipsparseScsrgemm2_bufferSizeExt;                                     \
      const auto &cusparsecsrgemm2 = hipsparseScsrgemm2;                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case torch::ScalarType::Double: {                                          \
      using scalar_t = double;                                                 \
      const auto &cusparsecsrgemm2_bufferSizeExt =                             \
          hipsparseDcsrgemm2_bufferSizeExt;                                     \
      const auto &cusparsecsrgemm2 = hipsparseDcsrgemm2;                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    default:                                                                   \
      AT_ERROR("Not implemented for '", toString(TYPE), "'");                  \
    }                                                                          \
  }()

std::tuple<torch::Tensor, torch::Tensor, torch::optional<torch::Tensor>>
spspmm_cuda(torch::Tensor rowptrA, torch::Tensor colA,
            torch::optional<torch::Tensor> optional_valueA,
            torch::Tensor rowptrB, torch::Tensor colB,
            torch::optional<torch::Tensor> optional_valueB, int64_t K,
            std::string reduce) {

  CHECK_CUDA(rowptrA);
  CHECK_CUDA(colA);
  if (optional_valueA.has_value())
    CHECK_CUDA(optional_valueA.value());
  CHECK_CUDA(rowptrB);
  CHECK_CUDA(colB);
  if (optional_valueB.has_value())
    CHECK_CUDA(optional_valueB.value());
  hipSetDevice(rowptrA.get_device());

  CHECK_INPUT(rowptrA.dim() == 1);
  CHECK_INPUT(colA.dim() == 1);
  if (optional_valueA.has_value()) {
    CHECK_INPUT(optional_valueA.value().dim() == 1);
    CHECK_INPUT(optional_valueA.value().size(0) == colA.size(0));
  }
  CHECK_INPUT(rowptrB.dim() == 1);
  CHECK_INPUT(colB.dim() == 1);
  if (optional_valueB.has_value()) {
    CHECK_INPUT(optional_valueB.value().dim() == 1);
    CHECK_INPUT(optional_valueB.value().size(0) == colB.size(0));
  }

  if (!optional_valueA.has_value() && optional_valueB.has_value())
    optional_valueA =
        torch::ones({colA.numel()}, optional_valueB.value().options());

  if (!optional_valueB.has_value() && optional_valueA.has_value())
    optional_valueB =
        torch::ones({colB.numel()}, optional_valueA.value().options());

  auto scalar_type = torch::ScalarType::Float;
  if (optional_valueA.has_value())
    scalar_type = optional_valueA.value().scalar_type();

  auto handle = at::cuda::getCurrentCUDASparseHandle();

  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);

  rowptrA = rowptrA.toType(torch::kInt);
  colA = colA.toType(torch::kInt);
  rowptrB = rowptrB.toType(torch::kInt);
  colB = colB.toType(torch::kInt);

  int64_t M = rowptrA.numel() - 1, N = rowptrB.numel() - 1;
  auto rowptrA_data = rowptrA.data_ptr<int>();
  auto colA_data = colA.data_ptr<int>();
  auto rowptrB_data = rowptrB.data_ptr<int>();
  auto colB_data = colB.data_ptr<int>();

  torch::Tensor rowptrC, colC;
  torch::optional<torch::Tensor> optional_valueC = torch::nullopt;

  int nnzC;
  int *nnzTotalDevHostPtr = &nnzC;

  // Step 1: Create an opaque structure.
  csrgemm2Info_t info = NULL;
  hipsparseCreateCsrgemm2Info(&info);

  // Step 2: Allocate buffer for `csrgemm2Nnz` and `csrgemm2`.
  size_t bufferSize;
  AT_DISPATCH_CUSPARSE_TYPES(scalar_type, [&] {
    scalar_t alpha = (scalar_t)1.0;
    cusparsecsrgemm2_bufferSizeExt(handle, M, N, K, &alpha, descr, colA.numel(),
                                   rowptrA_data, colA_data, descr, colB.numel(),
                                   rowptrB_data, colB_data, NULL, descr, 0,
                                   NULL, NULL, info, &bufferSize);

    void *buffer = NULL;
    hipMalloc(&buffer, bufferSize);

    // Step 3: Compute CSR row pointer.
    rowptrC = torch::empty({M + 1}, rowptrA.options());
    auto rowptrC_data = rowptrC.data_ptr<int>();
    hipsparseXcsrgemm2Nnz(handle, M, N, K, descr, colA.numel(), rowptrA_data,
                         colA_data, descr, colB.numel(), rowptrB_data,
                         colB_data, descr, 0, NULL, NULL, descr, rowptrC_data,
                         nnzTotalDevHostPtr, info, buffer);

    // Step 4: Compute CSR entries.
    colC = torch::empty({nnzC}, rowptrC.options());
    auto colC_data = colC.data_ptr<int>();

    if (optional_valueA.has_value())
      optional_valueC = torch::empty({nnzC}, optional_valueA.value().options());

    scalar_t *valA_data = NULL, *valB_data = NULL, *valC_data = NULL;
    if (optional_valueA.has_value()) {
      valA_data = optional_valueA.value().data_ptr<scalar_t>();
      valB_data = optional_valueB.value().data_ptr<scalar_t>();
      valC_data = optional_valueC.value().data_ptr<scalar_t>();
    }

    cusparsecsrgemm2(handle, M, N, K, &alpha, descr, colA.numel(), valA_data,
                     rowptrA_data, colA_data, descr, colB.numel(), valB_data,
                     rowptrB_data, colB_data, NULL, descr, 0, NULL, NULL, NULL,
                     descr, valC_data, rowptrC_data, colC_data, info, buffer);

    hipFree(buffer);
  });

  // Step 5: Destroy the opaque structure.
  hipsparseDestroyCsrgemm2Info(info);

  rowptrC = rowptrC.toType(torch::kLong);
  colC = colC.toType(torch::kLong);

  return std::make_tuple(rowptrC, colC, optional_valueC);
}
