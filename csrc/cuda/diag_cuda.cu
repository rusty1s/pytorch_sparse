#include "hip/hip_runtime.h"
#include "diag_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024

__global__ void non_diag_mask_kernel(const int64_t *row_data,
                                     const int64_t *col_data, bool *out_data,
                                     int64_t N, int64_t k, int64_t num_diag,
                                     int64_t numel) {

  int64_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (thread_idx < numel) {
    int64_t r = row_data[thread_idx], c = col_data[thread_idx];

    if (k < 0) {
      if (r + k < 0) {
        out_data[thread_idx] = true;
      } else if (r + k >= N) {
        out_data[thread_idx + num_diag] = true;
      } else if (r + k > c) {
        out_data[thread_idx + r + k] = true;
      } else if (r + k < c) {
        out_data[thread_idx + r + k + 1] = true;
      }

    } else {
      if (r + k >= N) {
        out_data[thread_idx + num_diag] = true;
      } else if (r + k > c) {
        out_data[thread_idx + r] = true;
      } else if (r + k < c) {
        out_data[thread_idx + r + 1] = true;
      }
    }
  }
}

torch::Tensor non_diag_mask_cuda(torch::Tensor row, torch::Tensor col,
                                 int64_t M, int64_t N, int64_t k) {
  CHECK_CUDA(row);
  CHECK_CUDA(col);
  hipSetDevice(row.get_device());

  auto E = row.size(0);
  auto num_diag = k < 0 ? std::min(M + k, N) : std::min(M, N - k);

  auto row_data = row.data_ptr<int64_t>();
  auto col_data = col.data_ptr<int64_t>();

  auto mask = torch::zeros({E + num_diag}, row.options().dtype(torch::kBool));
  auto mask_data = mask.data_ptr<bool>();

  if (E == 0)
    return mask;

  auto stream = at::cuda::getCurrentCUDAStream();
  non_diag_mask_kernel<<<(E + THREADS - 1) / THREADS, THREADS, 0, stream>>>(
      row_data, col_data, mask_data, N, k, num_diag, E);

  return mask;
}
